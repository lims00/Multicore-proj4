
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#include <sys/time.h>
// 추가된 CUDA 관련 헤더 파일
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std::chrono;

#define CUDA 0
#define OPENMP 1
#define SPHERES 20
#define DIM 2048

#define rnd(x) (x * rand() / RAND_MAX)
#define INF 2e10f

struct Sphere {
    float r, b, g;
    float radius;
    float x, y, z;
    float (*hit)(struct Sphere* s, float ox, float oy, float* n);
};

// CUDA 커널 함수 추가
__device__ float hit(struct Sphere* s, float ox, float oy, float* n) {
    float dx = ox - s->x;
    float dy = oy - s->y;
    if (dx * dx + dy * dy < s->radius * s->radius) {
        float dz = sqrtf(s->radius * s->radius - dx * dx - dy * dy);
        *n = dz / sqrtf(s->radius * s->radius);
        return dz + s->z;
    }
    return -INF;
}

// CUDA 커널 함수 추가
__global__ void kernel(struct Sphere* s, unsigned char* ptr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * DIM;
    float ox = (x - DIM / 2);
    float oy = (y - DIM / 2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for (int i = 0; i < SPHERES; i++) {
        float n;
        float t = hit(&s[i], ox, oy, &n);
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}

void ppm_write(unsigned char* bitmap, int xdim, int ydim, FILE* fp) {
    int i, x, y;
    fprintf(fp, "P3\n");
    fprintf(fp, "%d %d\n", xdim, ydim);
    fprintf(fp, "255\n");
    for (y = 0; y < ydim; y++) {
        for (x = 0; x < xdim; x++) {
            i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
        }
        fprintf(fp, "\n");
    }
}

int main(void) {
    int no_threads;
    int option;
    int x, y;
    unsigned char* bitmap;
    
    
    srand(time(NULL));

    FILE* fp = fopen("cudaresult", "w");

    

    struct Sphere* temp_s = (struct Sphere*)malloc(sizeof(struct Sphere) * SPHERES);
    for (int i = 0; i < SPHERES; i++) {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(2000.0f) - 1000;
        temp_s[i].y = rnd(2000.0f) - 1000;
        temp_s[i].z = rnd(2000.0f) - 1000;
        temp_s[i].radius = rnd(200.0f) + 40;
    }

    bitmap = (unsigned char*)malloc(sizeof(unsigned char) * DIM * DIM * 4);
    auto start = high_resolution_clock::now();
           // GPU에서 사용할 메모리 할당
    struct Sphere* dev_s;
    unsigned char* dev_bitmap;
    hipMalloc((void**)&dev_s, sizeof(struct Sphere) * SPHERES);
    hipMalloc((void**)&dev_bitmap, sizeof(unsigned char) * DIM * DIM * 4);
    // CPU에서 GPU로 데이터 복사
    hipMemcpy(dev_s, temp_s, sizeof(struct Sphere) * SPHERES, hipMemcpyHostToDevice);
    // CUDA 커널 호출
    dim3 blocksPerGrid(DIM / 16, DIM / 16);
    dim3 threadsPerBlock(16, 16);
    kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_s, dev_bitmap);
    // GPU에서 결과 이미지 데이터를 CPU로 복사
    hipMemcpy(bitmap, dev_bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost);
    // GPU 메모리 해제
    hipFree(dev_s);
    hipFree(dev_bitmap);
    
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<milliseconds>(stop - start);
    printf("DEVICE(CUDA) execution time:%dms",(int)duration.count());
    ppm_write(bitmap, DIM, DIM, fp);
    fclose(fp);
    free(bitmap);
    free(temp_s);

    return 0;
}
